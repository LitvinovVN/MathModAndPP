#pragma once
#include <hip/hip_runtime.h>


// cuda-ядро для вывода одномерного массива в консоль
template<typename T>
__global__
void print_kernel(T* data, size_t indStart, size_t length)
{
    int th_i = blockIdx.x * blockDim.x + threadIdx.x;
    if (th_i == 0)
    {
        //printf("GPU: print_kernel() vectorGpu._size = %d\n", vectorGpu.GetSize());
        T* _dev_data_pointer = data;
        auto indEnd = indStart + length - 1;
        /*if(indEnd > vectorGpu.GetSize())
        {
            printf("Error! indEnd > vectorGpu.GetSize()\n");
            return;
        }*/

        printf("[%d..", (long)indStart);
        printf("%d]: ", (long)indEnd);
        for(size_t i = indStart; i <= indEnd; i++)
        {
            printf("%f ", _dev_data_pointer[i]);
        }
        printf("\n");
    }
}


template<typename T>
__global__ void kernel_sum(T* dev_arr, size_t length, T* dev_block_sum, T* result)
{
    // Массив в распределенной памяти GPU
    // для хранения локальных сумм отдельных потоков блока
    extern __shared__ T shared_array[];

    //printf("\nkernel_sum: length = %ld\n", length);
    const int tid = threadIdx.x + blockDim.x * blockIdx.x;
    //printf("\nkernel_sum: tid = %ld\n", tid);    
    const int number_of_threads = gridDim.x * blockDim.x;
    int n_elem_per_thread = length / number_of_threads;
        
    #ifdef DEBUG
    if(tid == 0)
    {
        printf("\nkernel_sum: number_of_threads = %d\n", number_of_threads);
        printf("\nkernel_sum: n_elem_per_thread = %d\n", n_elem_per_thread);
    }
    #endif

    unsigned long long block_start_idx = n_elem_per_thread * blockIdx.x * blockDim.x;
    unsigned long long thread_start_idx = block_start_idx
            + threadIdx.x * n_elem_per_thread;
    unsigned long long thread_end_idx = thread_start_idx + n_elem_per_thread;
    if(tid == number_of_threads - 1)
    {
        thread_end_idx = length;
    }

    if(thread_end_idx > length) thread_end_idx = length;
    
    #ifdef DEBUG
    printf("\nkernel_sum: i = %d [%d .. %d]\n", tid, thread_start_idx, thread_end_idx);
    #endif

    T localResult{0};
    
    for(size_t i = thread_start_idx; i < thread_end_idx; i++)
    {
        localResult += dev_arr[i];
    }

    #ifdef DEBUG    
    printf("\nkernel_sum: i = %d, localResult = %f\n", tid, localResult);
    #endif
    
    shared_array[threadIdx.x] = localResult;
    __syncthreads();

    // Просматриваем содержимое распределяемой памяти
    #ifdef DEBUG
    if(threadIdx.x == 0)
    {
        for(int i = 0; i < blockDim.x; i++)
        {
            printf("\nkernel_sum: %d (b%d, t%d) shared_array[%d] = %f\n", tid, blockIdx.x, threadIdx.x, i, shared_array[i]);
        }
    }
    #endif
    
    if(threadIdx.x == 0)
    {
        T block_result = 0;
        for(int i = 0; i < blockDim.x; i++)
        {
            block_result += shared_array[i];
            #ifdef DEBUG
            printf("\nkernel_sum: shared_array[%d] = %f\n", tid, shared_array[i]);
            #endif
        }
        #ifdef DEBUG
        printf("\nkernel_sum: %d, block_result = %f\n", tid, block_result);
        #endif
        dev_block_sum[blockIdx.x] = block_result;
    }
    
}
