// Задача 01. Вывести с помощью GPU текстовую строку "Hello World from GPU!"
// Запуск:
// nvcc kernel2.cu -o app --run
// ./app


#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

__host__ __device__
void printArray(float* arr_GPU, int numElements)
{
    for(int i = 0; i < numElements; i++)
    {
        printf("%g ", arr_GPU[i]);
    }
    printf("\n"); 
}

__host__ __device__
void testCalculation(float* arr, int numElements)
{
    for(int i = 1; i < numElements-1; i++)
    {
        arr[i] = - 1.5f * arr[i-1] + 2.0f * arr[i] - arr[i+1] + 10;
    }  
}


__global__ void cuda_print_array(float* arr_GPU, int numElements){
    printf("Hello World from GPU!\n");

    printArray(arr_GPU, numElements);
    printf("\n");

    testCalculation(arr_GPU, numElements);
    printArray(arr_GPU, numElements);
}

__global__ void cuda_calculation(float* arr_GPU_01, float* arr_GPU_02, float* arr_GPU_03, float* arr_GPU_04,
    float* arr_GPU_05, float* arr_GPU_06, float* arr_GPU_07, float* arr_GPU_08,
    float* arr_GPU_09, float* arr_GPU_10, float* arr_GPU_11, float* arr_GPU_12,
    float* arr_GPU_13, float* arr_GPU_14, float* arr_GPU_15, float* arr_GPU_16,
    float* arr_GPU_17, float* arr_GPU_18, float* arr_GPU_19, float* arr_GPU_20,
    float* arr_GPU_21, float* arr_GPU_22, float* arr_GPU_23, float* arr_GPU_24,
    float* arr_GPU_25, float* arr_GPU_26, float* arr_GPU_27, float* arr_GPU_28,
    int numElements){
    //printf("---cuda_calculation---\n");
    float* calculatingArr = NULL;
    int blockIndex = blockIdx.x;
    if(blockIndex == 0) calculatingArr = arr_GPU_01;
    if(blockIndex == 1) calculatingArr = arr_GPU_02;
    if(blockIndex == 2) calculatingArr = arr_GPU_03;
    if(blockIndex == 3) calculatingArr = arr_GPU_04;
    if(blockIndex == 4) calculatingArr = arr_GPU_05;
    if(blockIndex == 5) calculatingArr = arr_GPU_06;
    if(blockIndex == 6) calculatingArr = arr_GPU_07;
    if(blockIndex == 7) calculatingArr = arr_GPU_08;
    if(blockIndex == 8) calculatingArr = arr_GPU_09;
    if(blockIndex == 9) calculatingArr = arr_GPU_10;
    if(blockIndex == 10) calculatingArr = arr_GPU_11;
    if(blockIndex == 11) calculatingArr = arr_GPU_12;
    if(blockIndex == 12) calculatingArr = arr_GPU_13;
    if(blockIndex == 13) calculatingArr = arr_GPU_14;
    if(blockIndex == 14) calculatingArr = arr_GPU_15;
    if(blockIndex == 15) calculatingArr = arr_GPU_16;
    if(blockIndex == 16) calculatingArr = arr_GPU_17;
    if(blockIndex == 17) calculatingArr = arr_GPU_18;
    if(blockIndex == 18) calculatingArr = arr_GPU_19;
    if(blockIndex == 19) calculatingArr = arr_GPU_20;
    if(blockIndex == 20) calculatingArr = arr_GPU_21;
    if(blockIndex == 21) calculatingArr = arr_GPU_22;
    if(blockIndex == 22) calculatingArr = arr_GPU_23;
    if(blockIndex == 23) calculatingArr = arr_GPU_24;
    if(blockIndex == 24) calculatingArr = arr_GPU_25;
    if(blockIndex == 25) calculatingArr = arr_GPU_26;
    if(blockIndex == 26) calculatingArr = arr_GPU_27;
    if(blockIndex == 27) calculatingArr = arr_GPU_28;
    
    if (calculatingArr != NULL) testCalculation(calculatingArr, numElements);
    //testCalculation(arr_GPU_01, numElements);
    //testCalculation(arr_GPU_02, numElements);
}

int main() {
    int numElements = 10000000;
    size_t dataSize = numElements * sizeof(float);
    float* arr_RAM = (float*)malloc(dataSize);
    for(int i = 0; i < numElements; i++)
    {
        arr_RAM[i] = i;
    }
    
    /////// CPU ///////
    float* arr_RAM_res = (float*)malloc(dataSize);
    for(int i = 0; i < numElements; i++)
    {
        arr_RAM_res[i] = arr_RAM[i];
    }

    clock_t t;
    t = clock();
    testCalculation(arr_RAM_res, numElements);
    //testCalculation(arr_RAM_res, numElements);
    t = clock() - t;
    double time_taken = ((double)t)*1000/CLOCKS_PER_SEC; // in milliseconds
 
    printf("CPU testCalculation() took %f milliseconds to execute \n", time_taken);
    ///////////////////


    float* arr_GPU_01;
    hipMalloc((void**)&arr_GPU_01, dataSize);
    hipMemcpy(arr_GPU_01, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_02;
    hipMalloc((void**)&arr_GPU_02, dataSize);
    hipMemcpy(arr_GPU_02, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_03;
    hipMalloc((void**)&arr_GPU_03, dataSize);
    hipMemcpy(arr_GPU_03, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_04;
    hipMalloc((void**)&arr_GPU_04, dataSize);
    hipMemcpy(arr_GPU_04, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_05;
    hipMalloc((void**)&arr_GPU_05, dataSize);
    hipMemcpy(arr_GPU_05, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_06;
    hipMalloc((void**)&arr_GPU_06, dataSize);
    hipMemcpy(arr_GPU_06, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_07;
    hipMalloc((void**)&arr_GPU_07, dataSize);
    hipMemcpy(arr_GPU_07, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_08;
    hipMalloc((void**)&arr_GPU_08, dataSize);
    hipMemcpy(arr_GPU_08, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_09;
    hipMalloc((void**)&arr_GPU_09, dataSize);
    hipMemcpy(arr_GPU_09, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_10;
    hipMalloc((void**)&arr_GPU_10, dataSize);
    hipMemcpy(arr_GPU_10, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_11;
    hipMalloc((void**)&arr_GPU_11, dataSize);
    hipMemcpy(arr_GPU_11, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_12;
    hipMalloc((void**)&arr_GPU_12, dataSize);
    hipMemcpy(arr_GPU_12, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_13;
    hipMalloc((void**)&arr_GPU_13, dataSize);
    hipMemcpy(arr_GPU_13, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_14;
    hipMalloc((void**)&arr_GPU_14, dataSize);
    hipMemcpy(arr_GPU_14, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_15;
    hipMalloc((void**)&arr_GPU_15, dataSize);
    hipMemcpy(arr_GPU_15, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_16;
    hipMalloc((void**)&arr_GPU_16, dataSize);
    hipMemcpy(arr_GPU_16, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_17;
    hipMalloc((void**)&arr_GPU_17, dataSize);
    hipMemcpy(arr_GPU_17, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_18;
    hipMalloc((void**)&arr_GPU_18, dataSize);
    hipMemcpy(arr_GPU_18, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_19;
    hipMalloc((void**)&arr_GPU_19, dataSize);
    hipMemcpy(arr_GPU_19, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_20;
    hipMalloc((void**)&arr_GPU_20, dataSize);
    hipMemcpy(arr_GPU_20, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_21;
    hipMalloc((void**)&arr_GPU_21, dataSize);
    hipMemcpy(arr_GPU_21, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_22;
    hipMalloc((void**)&arr_GPU_22, dataSize);
    hipMemcpy(arr_GPU_22, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_23;
    hipMalloc((void**)&arr_GPU_23, dataSize);
    hipMemcpy(arr_GPU_23, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_24;
    hipMalloc((void**)&arr_GPU_24, dataSize);
    hipMemcpy(arr_GPU_24, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_25;
    hipMalloc((void**)&arr_GPU_25, dataSize);
    hipMemcpy(arr_GPU_25, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_26;
    hipMalloc((void**)&arr_GPU_26, dataSize);
    hipMemcpy(arr_GPU_26, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_27;
    hipMalloc((void**)&arr_GPU_27, dataSize);
    hipMemcpy(arr_GPU_27, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_28;
    hipMalloc((void**)&arr_GPU_28, dataSize);
    hipMemcpy(arr_GPU_28, arr_RAM, dataSize, hipMemcpyHostToDevice);

    // Prepare
    hipStream_t stream1, stream2, stream3, stream4;
    hipStream_t stream5, stream6, stream7, stream8;
    hipStream_t stream9, stream10, stream11, stream12;
    hipStream_t stream13, stream14, stream15, stream16;
    hipStreamCreate (&stream1);
    hipStreamCreate (&stream2);
    hipStreamCreate (&stream3);
    hipStreamCreate (&stream4);
    hipStreamCreate (&stream5);
    hipStreamCreate (&stream6);
    hipStreamCreate (&stream7);
    hipStreamCreate (&stream8);
    hipStreamCreate (&stream9);
    hipStreamCreate (&stream10);
    hipStreamCreate (&stream11);
    hipStreamCreate (&stream12);
    hipStreamCreate (&stream13);
    hipStreamCreate (&stream14);
    hipStreamCreate (&stream15);
    hipStreamCreate (&stream16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);
    // Do something on GPU
    cuda_calculation<<<28,1,0,stream1>>>(arr_GPU_01, arr_GPU_02, arr_GPU_03, arr_GPU_04,
        arr_GPU_05, arr_GPU_06, arr_GPU_07, arr_GPU_08,
        arr_GPU_09, arr_GPU_10, arr_GPU_11, arr_GPU_12,
        arr_GPU_13, arr_GPU_14, arr_GPU_15, arr_GPU_16,
        arr_GPU_17, arr_GPU_18, arr_GPU_19, arr_GPU_20,
        arr_GPU_21, arr_GPU_22, arr_GPU_23, arr_GPU_24,
        arr_GPU_25, arr_GPU_26, arr_GPU_27, arr_GPU_28,
        numElements);
    //cuda_calculation<<<1,1,0,stream2>>>(arr_GPU_02, arr_GPU_02, numElements);
    //cuda_calculation<<<1,1,0,stream3>>>(arr_GPU_03, arr_GPU_03, numElements);
    //cuda_calculation<<<1,1,0,stream4>>>(arr_GPU_04, arr_GPU_04, numElements);
    //cuda_calculation<<<1,1,0,stream5>>>(arr_GPU_05, arr_GPU_05, numElements);
    //cuda_calculation<<<1,1,0,stream6>>>(arr_GPU_06, arr_GPU_06, numElements);
    //cuda_calculation<<<1,1,0,stream7>>>(arr_GPU_07, arr_GPU_07, numElements);
    //cuda_calculation<<<1,1,0,stream8>>>(arr_GPU_08, arr_GPU_08, numElements);
    //cuda_calculation<<<1,1,0,stream9>>>(arr_GPU_09, arr_GPU_09, numElements);
    //cuda_calculation<<<1,1,0,stream10>>>(arr_GPU_10, arr_GPU_10, numElements);
    //cuda_calculation<<<1,1,0,stream11>>>(arr_GPU_11, arr_GPU_11, numElements);
    //cuda_calculation<<<1,1,0,stream12>>>(arr_GPU_12, arr_GPU_12, numElements);
    //cuda_calculation<<<1,1,0,stream13>>>(arr_GPU_13, arr_GPU_13, numElements);
    //cuda_calculation<<<1,1,0,stream14>>>(arr_GPU_14, arr_GPU_14, numElements);
    //cuda_calculation<<<1,1,0,stream15>>>(arr_GPU_15, arr_GPU_15, numElements);
    //cuda_calculation<<<1,1,0,stream16>>>(arr_GPU_16, arr_GPU_16, numElements);    
    hipDeviceSynchronize ();    
    // Stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("\nelapsedTime GPU = %g", elapsedTime);


    //////////// Сравнение результатов CPU и GPU ///////////
    float* arr_GPU_res1 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res1, arr_GPU_01, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res2 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res2, arr_GPU_02, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res3 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res3, arr_GPU_03, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res4 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res4, arr_GPU_04, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res5 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res5, arr_GPU_05, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res6 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res6, arr_GPU_06, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res7 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res7, arr_GPU_07, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res8 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res8, arr_GPU_08, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res9 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res9, arr_GPU_09, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res10 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res10, arr_GPU_10, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res11 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res11, arr_GPU_11, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res12 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res12, arr_GPU_12, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res13 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res13, arr_GPU_13, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res14 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res14, arr_GPU_14, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res15 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res15, arr_GPU_15, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res16 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res16, arr_GPU_16, dataSize, hipMemcpyDeviceToHost);
/*
    for(int i = 0; i < numElements; i++)
    {
        if(abs(arr_GPU_res1[i] - arr_RAM_res[i]) > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res1[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 1 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res2[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res2[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 2 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res3[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res3[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 3 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res4[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res4[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 4 OK------");  

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res5[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res5[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 5 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res6[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res6[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 6 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res7[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res7[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 7 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res8[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res8[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 8 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res9[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res9[i], arr_RAM_res[i]);
    }
*/


    printf("\n----TESTS OK------\n");

    //printArray(arr_RAM_res, numElements);
    //printf("\n----TESTS OK------\n");
    //printArray(arr_GPU_res1, numElements);
    //printf("\n----TESTS OK------\n");
    //printArray(arr_GPU_res2, numElements);
    ////////////////////////////////////////////////////////


    return 0;
}