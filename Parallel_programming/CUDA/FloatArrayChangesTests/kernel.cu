// Задача 01. Вывести с помощью GPU текстовую строку "Hello World from GPU!"
// Запуск:
// nvcc kernel.cu -o app
// ./app


#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

__host__ __device__
void printArray(float* arr_GPU, int numElements)
{
    for(int i = 0; i < numElements; i++)
    {
        printf("%g ", arr_GPU[i]);
    }
    printf("\n"); 
}

__host__ __device__
void testCalculation(float* arr, int numElements)
{
    for(int i = 1; i < numElements-1; i++)
    {
        arr[i] = - arr[i-1] + 2 * arr[i] - arr[i+1] + 10;
    }  
}


__global__ void cuda_print_array(float* arr_GPU, int numElements){
    printf("Hello World from GPU!\n");

    printArray(arr_GPU, numElements);
    printf("\n");

    testCalculation(arr_GPU, numElements);
    printArray(arr_GPU, numElements);
}

__global__ void cuda_calculation(float* arr_GPU_01, float* arr_GPU_02, int numElements){
    //printf("---cuda_calculation---\n");
    testCalculation(arr_GPU_01, numElements);
    //testCalculation(arr_GPU_02, numElements);
}

int main() {
    int numElements = 1000000;
    size_t dataSize = numElements * sizeof(float);
    float* arr_RAM = (float*)malloc(dataSize);
    for(int i = 0; i < numElements; i++)
    {
        arr_RAM[i] = i;
    }
    
    /////// CPU ///////
    float* arr_RAM_res = (float*)malloc(dataSize);
    for(int i = 0; i < numElements; i++)
    {
        arr_RAM_res[i] = arr_RAM[i];
    }

    clock_t t;
    t = clock();
    testCalculation(arr_RAM_res, numElements);
    //testCalculation(arr_RAM_res, numElements);
    t = clock() - t;
    double time_taken = ((double)t)*1000/CLOCKS_PER_SEC; // in milliseconds
 
    printf("CPU testCalculation() took %f milliseconds to execute \n", time_taken);
    ///////////////////


    float* arr_GPU_01;
    hipMalloc((void**)&arr_GPU_01, dataSize);
    hipMemcpy(arr_GPU_01, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_02;
    hipMalloc((void**)&arr_GPU_02, dataSize);
    hipMemcpy(arr_GPU_02, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_03;
    hipMalloc((void**)&arr_GPU_03, dataSize);
    hipMemcpy(arr_GPU_03, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_04;
    hipMalloc((void**)&arr_GPU_04, dataSize);
    hipMemcpy(arr_GPU_04, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_05;
    hipMalloc((void**)&arr_GPU_05, dataSize);
    hipMemcpy(arr_GPU_05, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_06;
    hipMalloc((void**)&arr_GPU_06, dataSize);
    hipMemcpy(arr_GPU_06, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_07;
    hipMalloc((void**)&arr_GPU_07, dataSize);
    hipMemcpy(arr_GPU_07, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_08;
    hipMalloc((void**)&arr_GPU_08, dataSize);
    hipMemcpy(arr_GPU_08, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_09;
    hipMalloc((void**)&arr_GPU_09, dataSize);
    hipMemcpy(arr_GPU_09, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_10;
    hipMalloc((void**)&arr_GPU_10, dataSize);
    hipMemcpy(arr_GPU_10, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_11;
    hipMalloc((void**)&arr_GPU_11, dataSize);
    hipMemcpy(arr_GPU_11, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_12;
    hipMalloc((void**)&arr_GPU_12, dataSize);
    hipMemcpy(arr_GPU_12, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_13;
    hipMalloc((void**)&arr_GPU_13, dataSize);
    hipMemcpy(arr_GPU_13, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_14;
    hipMalloc((void**)&arr_GPU_14, dataSize);
    hipMemcpy(arr_GPU_14, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_15;
    hipMalloc((void**)&arr_GPU_15, dataSize);
    hipMemcpy(arr_GPU_15, arr_RAM, dataSize, hipMemcpyHostToDevice);

    float* arr_GPU_16;
    hipMalloc((void**)&arr_GPU_16, dataSize);
    hipMemcpy(arr_GPU_16, arr_RAM, dataSize, hipMemcpyHostToDevice);

    // Prepare
    hipStream_t stream1, stream2, stream3, stream4;
    hipStream_t stream5, stream6, stream7, stream8;
    hipStream_t stream9, stream10, stream11, stream12;
    hipStream_t stream13, stream14, stream15, stream16;
    hipStreamCreate (&stream1);
    hipStreamCreate (&stream2);
    hipStreamCreate (&stream3);
    hipStreamCreate (&stream4);
    hipStreamCreate (&stream5);
    hipStreamCreate (&stream6);
    hipStreamCreate (&stream7);
    hipStreamCreate (&stream8);
    hipStreamCreate (&stream9);
    hipStreamCreate (&stream10);
    hipStreamCreate (&stream11);
    hipStreamCreate (&stream12);
    hipStreamCreate (&stream13);
    hipStreamCreate (&stream14);
    hipStreamCreate (&stream15);
    hipStreamCreate (&stream16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);
    // Do something on GPU
    cuda_calculation<<<1,1,0,stream1>>>(arr_GPU_01, arr_GPU_01, numElements);
    cuda_calculation<<<1,1,0,stream2>>>(arr_GPU_02, arr_GPU_02, numElements);
    cuda_calculation<<<1,1,0,stream3>>>(arr_GPU_03, arr_GPU_03, numElements);
    cuda_calculation<<<1,1,0,stream4>>>(arr_GPU_04, arr_GPU_04, numElements);
    cuda_calculation<<<1,1,0,stream5>>>(arr_GPU_05, arr_GPU_05, numElements);
    cuda_calculation<<<1,1,0,stream6>>>(arr_GPU_06, arr_GPU_06, numElements);
    cuda_calculation<<<1,1,0,stream7>>>(arr_GPU_07, arr_GPU_07, numElements);
    cuda_calculation<<<1,1,0,stream8>>>(arr_GPU_08, arr_GPU_08, numElements);
    //cuda_calculation<<<1,1,0,stream9>>>(arr_GPU_09, arr_GPU_09, numElements);
    //cuda_calculation<<<1,1,0,stream10>>>(arr_GPU_10, arr_GPU_10, numElements);
    //cuda_calculation<<<1,1,0,stream11>>>(arr_GPU_11, arr_GPU_11, numElements);
    //cuda_calculation<<<1,1,0,stream12>>>(arr_GPU_12, arr_GPU_12, numElements);
    //cuda_calculation<<<1,1,0,stream13>>>(arr_GPU_13, arr_GPU_13, numElements);
    //cuda_calculation<<<1,1,0,stream14>>>(arr_GPU_14, arr_GPU_14, numElements);
    //cuda_calculation<<<1,1,0,stream15>>>(arr_GPU_15, arr_GPU_15, numElements);
    //cuda_calculation<<<1,1,0,stream16>>>(arr_GPU_16, arr_GPU_16, numElements);    
    hipDeviceSynchronize ();    
    // Stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("\nelapsedTime GPU = %g", elapsedTime);


    //////////// Сравнение результатов CPU и GPU ///////////
    float* arr_GPU_res1 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res1, arr_GPU_01, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res2 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res2, arr_GPU_02, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res3 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res3, arr_GPU_03, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res4 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res4, arr_GPU_04, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res5 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res5, arr_GPU_05, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res6 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res6, arr_GPU_06, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res7 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res7, arr_GPU_07, dataSize, hipMemcpyDeviceToHost);
    float* arr_GPU_res8 = (float*)malloc(dataSize);
    hipMemcpy(arr_GPU_res8, arr_GPU_08, dataSize, hipMemcpyDeviceToHost);

    for(int i = 0; i < numElements; i++)
    {
        if(abs(arr_GPU_res1[i] - arr_RAM_res[i]) > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res1[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 1 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res2[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res2[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 2 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res3[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res3[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 3 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res4[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res4[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 4 OK------");  

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res5[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res5[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 5 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res6[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res6[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 6 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res7[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res7[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 7 OK------");

    for(int i = 0; i < numElements; i++)
    {
        if(arr_GPU_res8[i] - arr_RAM_res[i] > 0.000001)
            printf("ERROR! i=%d %g %g\n", i, arr_GPU_res8[i], arr_RAM_res[i]);
    }
    //printf("\n----TEST 8 OK------");

    printf("\n----TESTS OK------");
    ////////////////////////////////////////////////////////


    return 0;
}