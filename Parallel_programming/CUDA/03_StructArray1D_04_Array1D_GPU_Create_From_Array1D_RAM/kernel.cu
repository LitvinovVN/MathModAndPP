/* Задача 03_04. Добавить функцию Array1D_GPU_Create_From_Array1D_RAM,
 создающую в GPU структуру Array1D на основе структуры Array1D, размещённой в ОЗУ

 Запуск:
 nvcc kernel.cu -o app.exe
 ./app

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> // Для использования malloc

///////////////////////////////////////////////////////

// Получает строку-сообщение message, выводит её в консоль.
// Считывает целое число, введённое пользователем и возвращает его.
int IntNumber_Input(const char message[])
{
    int numElements;
    printf(message);
    scanf("%d", &numElements);

    return numElements;
}
///////////////////////////////////////////////////////

// Создаёт массив float в динамической памяти и возвращает на него указатель
float* FloatArray_RAM_Create(int numElements)
{
    float* arr = (float*)malloc(numElements * sizeof(float));
    return arr;
}

//  Создаёт массив float в GPU и возвращает на него указатель
float* FloatArray_GPU_Create(int numElements)
{
    float* arrayGPU;
    hipMalloc((void**)&arrayGPU, numElements * sizeof(float));
    return arrayGPU;
}

// Копирует элементы массива fArray_RAM, расположенного в ОЗУ, в массив fArray_GPU, расположенный в видеопамяти.
void FloatArray_CopyFromRAMtoGPU(float* fArray_RAM, float* fArray_GPU, int numElements)
{
    hipMemcpy(fArray_GPU, fArray_RAM, numElements * sizeof(float), hipMemcpyHostToDevice);
}

// Копирует элементы массива fArray_GPU, расположенного в видеопамяти, в массив fArray_RAM, расположенный в ОЗУ.
void FloatArray_CopyFromGPUtoRAM(float* fArray_GPU, float* fArray_RAM, int numElements)
{
    hipMemcpy(fArray_RAM, fArray_GPU, numElements * sizeof(float), hipMemcpyDeviceToHost);
}

//////////////////////////////////////////////////////////////////////////////

// Инициализирует элементы массива их индексами
__host__ __device__
void FloatArray_InitByIndexes(float* fArray, int numElements)
{
    size_t i = 0;
    while(i < numElements)
    {
        fArray[i] = i;
        i++;
    }        
}

// Выводит элементы массива в консоль
__host__ __device__
void FloatArray_Print(float* fArray, int numElements)
{
    size_t i = 0;
    while(i < numElements)
    {
        printf("%g ",fArray[i]);
        i++;
    }
    printf("\n");      
}

///////////////////////////////////////////////////////
// Создаем структуру "Одномерный массив"
struct array_t
{
    int size;
    float* data;
};

// Определяем новый тип
typedef struct array_t Array1D;
/////////////////////////////////////////////////////// 

// Создаёт структуру типа Array1D в ОЗУ
Array1D Array1D_RAM_Create(int numElements)
{
    Array1D arr = {numElements};
    arr.data = (float*)malloc(numElements * sizeof(float));

    int i = 0;

    while(i < arr.size)
    {
        arr.data[i] = 0;
        i++;
    }
    return arr;
}


// Создаёт структуру типа Array1D в памяти GPU и возвращает на неё указатель
Array1D* Array1D_GPU_Create(int numElements)
{
    printf("Array1D_GPU_Create started\n");
    // 1. Выделяем память в GPU под структуру Array1D
    Array1D* array1D_GPU;    
    hipMalloc((void**)&array1D_GPU, sizeof(Array1D));
    
    // 2. Копируем numElements в поле array1D_GPU->size структуры Array1D, рассположенной в GPU
    hipMemcpy(&(array1D_GPU->size), &numElements, sizeof(array1D_GPU->size), hipMemcpyHostToDevice);
    
    // 3. Выделяем память в GPU для хранения массива, в котором количество элементов = numElements
    float* data_GPU = FloatArray_GPU_Create(numElements);
    
    // 4. Создаем массив data_RAM в ОЗУ и инициализируем элементы массива их индексами
    float* data_RAM = FloatArray_RAM_Create(numElements);
    FloatArray_InitByIndexes(data_RAM, numElements);       

    // 5. Копируем массив data_RAM из GPU в массив data_GPU в ОЗУ
    hipMemcpy(data_GPU, data_RAM, numElements * sizeof(*(array1D_GPU->data)), hipMemcpyHostToDevice);
    
    // 6. Копируем указатель на массив data_GPU в поле data массива array1D_GPU, находящегося на GPU
    hipMemcpy(&(array1D_GPU->data), &data_GPU, sizeof(array1D_GPU->data), hipMemcpyHostToDevice);

    printf("Array1D_GPU_Create ended\n");
    return array1D_GPU;
}


// Инициализирует элементы массива структуры Array1D их индексами
void Array1D_RAM_InitByIndexes(Array1D arr)
{
    size_t i = 0;
    while(i < arr.size)
    {
        arr.data[i] = i;
        i++;
    }        
}

// Выводит элементы массива в консоль
__host__ __device__
void Array1D_Print(Array1D* array1D)
{
    printf("size=%d\n",array1D->size);
    size_t i = 0;
    while(i < array1D->size)
    {
        //printf("i=%d: ",i);
        printf("%g ",array1D->data[i]);
        i++;
    }
    printf("\n");      
}

//////////////////////////////////////////////////////////////////////////////

// Выводит в консоль массив fArray_GPU, расположенный в видеопамяти и содержащий numElements элементов типа float
__global__ void CudaFloatArray_Print(float* fArray_GPU, int numElements)
{
    printf("CudaFloatArray_Print:\n");
    FloatArray_Print(fArray_GPU, numElements);    
}

// Выводит в консоль структуру Array1D, расположенную в GPU. 
__global__ void CudaArray1D_GPU_Print(Array1D* array1D_GPU)
{
    printf("CudaArray1D_GPU_Print:\n");
    Array1D_Print(array1D_GPU);    
}

// Прибавляет число number к каждому элементу массива arr->data, рсположенному в GPU
__global__ void CudaArray1D_AddNumber(Array1D* arr, float number)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    arr->data[index] = arr->data[index] + number; 
}


//////////////////////////////////////////////////////////////////////////////

/* Создаёт структуру типа Array1D в ОЗУ как копию структуры Array1D,
 расположенной в GPU, и возвращает на неё указатель */
 Array1D* Array1D_RAM_Create_From_Array1D_GPU(Array1D* array1D_GPU)
 {
    Array1D* array1D_RAM = (Array1D*)malloc(sizeof(Array1D));
 
    Array1D* array1D_DTO = (Array1D*)malloc(sizeof(Array1D));
    // Копируем структуру из GPU в ОЗУ 
    hipMemcpy(array1D_DTO, array1D_GPU, sizeof(Array1D), hipMemcpyDeviceToHost);
    
    array1D_RAM->size = array1D_DTO->size;    
    array1D_RAM->data = (float*)malloc(array1D_DTO->size * sizeof(float));
    hipMemcpy(array1D_RAM->data, array1D_DTO->data, array1D_DTO->size * sizeof(float), hipMemcpyDeviceToHost);
    return array1D_RAM;
 }

 /* Создаёт структуру типа Array1D в GPU как копию структуры Array1D,
 расположенной в RAM, и возвращает на неё указатель */
 Array1D* Array1D_GPU_Create_From_Array1D_RAM(Array1D array1D_RAM)
 {    
    Array1D* array1D_GPU = Array1D_GPU_Create(array1D_RAM.size);
 
    Array1D* array1D_DTO = (Array1D*)malloc(sizeof(Array1D));
    hipMemcpy(array1D_DTO, array1D_GPU, sizeof(Array1D), hipMemcpyDeviceToHost);
    
    hipMemcpy(array1D_DTO->data, array1D_RAM.data, array1D_DTO->size * sizeof(float), hipMemcpyHostToDevice);
    
    free(array1D_DTO);
    
    return array1D_GPU;
 }

//////////////////////////////////////////////////////////////////////////////

void Array1D_RAM_Destruct(Array1D* array1D_RAM)
{
    free(array1D_RAM->data);
    free(array1D_RAM);
}

void Array1D_GPU_Destruct(Array1D* array1D_GPU)
{
    Array1D* array1D_DTO = (Array1D*)malloc(sizeof(Array1D));
    hipMemcpy(array1D_DTO, array1D_GPU, sizeof(Array1D), hipMemcpyDeviceToHost);
    hipFree(array1D_DTO->data);
    hipFree(array1D_DTO);
    free(array1D_DTO);
}

/////////////////////////////////////////////////////////////////////////////

int main()
{    
    int numElements = IntNumber_Input("Input number of array elements: ");
    printf("numElements = %d\n", numElements);

    Array1D array1D_RAM = Array1D_RAM_Create(numElements);
    Array1D_RAM_InitByIndexes(array1D_RAM);

    Array1D* array1D_GPU = Array1D_GPU_Create_From_Array1D_RAM(array1D_RAM);
    CudaArray1D_GPU_Print<<<1,1>>>(array1D_GPU);

    // Освобождаем память
    Array1D_RAM_Destruct(&array1D_RAM);
    Array1D_GPU_Destruct(array1D_GPU);

    return 0;
}