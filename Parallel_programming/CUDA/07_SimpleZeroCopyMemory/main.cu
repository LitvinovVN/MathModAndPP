// Задача 07. Создать переменную counter = 0 в нуль-копируемой памяти.
// Изменять значения переменной counter в диапазоне от 1 до 5 с паузой 1 секунда в функции CPU.
// Выводить значения переменной counter, если она больше 0, с некоторой паузой в CUDA-ядре.
// Запуск:
// nvcc main.cu -o app
// ./app

#include <iostream>              // подключаем заголовочный файл iostream (содержит определение std::cout)
#include <thread>                // подключаем библиотеку для работы с потоками
#include <chrono>                // sleep_for
#include "hip/hip_runtime.h"

using namespace std::chrono_literals;// для использования единиц измерения времени (ms)

void thread_function(int* counter)                 
{
    std::cout << "Thread function: counter = " << *counter << std::endl;

    while(*counter < 5)
    {
        (*counter)++;        
        std::cout << "Thread function: counter = " << *counter << std::endl;
        std::this_thread::sleep_for(1000ms);
    }    
}


using clock_value_t = long long;

__device__ void cuda_sleep(clock_value_t sleep_cycles)
{
    clock_value_t start = clock64();
    clock_value_t cycles_elapsed;
    do { cycles_elapsed = clock64() - start; } 
    while (cycles_elapsed < sleep_cycles);
}

__global__
void cuda_kernel_function(int* dev_counter)
{
    printf("cuda_kernel_function started\n");

    while(true)    
    {
        if(*dev_counter < 1) continue;

        printf("dev_counter = %d\n", *dev_counter);
        cuda_sleep(300000000);

        if(*dev_counter >= 5) break;
    }

    printf("cuda_kernel_function stopped\n");
}

int main()
{
    hipEvent_t cuda_start, cuda_stop;
    hipEventCreate(&cuda_start);
    hipEventCreate(&cuda_stop);

    int *counter = NULL;
    hipHostAlloc((void**)&counter, sizeof(int), hipHostMallocMapped);

    int *dev_counter = NULL;
    hipHostGetDevicePointer(&dev_counter, counter, 0);

    std::cout << "Main thread: Starting new thread...\n";

    hipEventRecord(cuda_start, 0);
    auto start = std::chrono::high_resolution_clock::now();

    cuda_kernel_function<<<1,1>>>(dev_counter);    // cuda_kernel_function starts running

    std::thread t(&thread_function, counter);   // t starts running
    std::cout << "Main thread: New thread started!\n";

    hipDeviceSynchronize();
    t.join();   // main thread waits for the thread t to finish
    
    std::cout << "Main thread: Thread joined\n";

    auto end = std::chrono::high_resolution_clock::now();
    hipEventRecord(cuda_stop, 0);

    std::chrono::duration<double, std::milli> elapsed = end-start;
    hipEventSynchronize(cuda_start);
    hipEventSynchronize(cuda_stop);
    float cuda_elapsedTime;
    hipEventElapsedTime(&cuda_elapsedTime, cuda_start, cuda_stop);

    std::cout << "---------------" << std::endl;
    std::cout << "Waited " << elapsed.count() << " ms (std::chrono)\n";
    std::cout << "Waited " << cuda_elapsedTime << " ms (CUDA)\n";
    
    std::cout << "counter = " << *counter << std::endl;

    return 0;
}