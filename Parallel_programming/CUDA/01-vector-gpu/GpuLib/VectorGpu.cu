#include "hip/hip_runtime.h"
#pragma once

#include <iostream>
#include <chrono>

#include "CudaHelper.cu"
#include "FuncResultScalar.cpp"
#include "VectorCpu.cpp"

// cuda-ядро для вывода одномерного массива в консоль
template<typename T>
__global__
void print_kernel(T* data, size_t indStart, size_t length)
{
    int th_i = blockIdx.x * blockDim.x + threadIdx.x;
    if (th_i == 0)
    {
        //printf("GPU: print_kernel() vectorGpu._size = %d\n", vectorGpu.GetSize());
        T* _dev_data_pointer = data;
        auto indEnd = indStart + length - 1;
        /*if(indEnd > vectorGpu.GetSize())
        {
            printf("Error! indEnd > vectorGpu.GetSize()\n");
            return;
        }*/

        printf("[%d..", (long)indStart);
        printf("%d]: ", (long)indEnd);
        for(size_t i = indStart; i <= indEnd; i++)
        {
            printf("%f ", _dev_data_pointer[i]);
        }
        printf("\n");
    }
}

/// @brief Вектор (в GPU) 
/// @tparam T Тип элементов вектора
template<typename T = double>
class VectorGpu
{
    // Количество элементов вектора
    size_t _size = 0;
    // Указатель на массив в видеопамяти
    T* _dev_data = nullptr;
    // Флаг инициализации вектора
    // false - неинициализирован, true - инициализирован
    bool _isInitialized = false;

public:
    VectorGpu(size_t size) : _size(size)
    {
        //std::cout << "VectorGpu(size_t size) constructor started...\n";

        if (_size == 0)
        {
            std::string mes = "Cannot initialize vector of _size = 0";
            //std::cerr << mes << std::endl;
            throw std::logic_error(mes);
        }

        hipError_t cudaResult = hipMalloc(&_dev_data, size*sizeof(T));
        if (cudaResult != hipSuccess)
        {
            std::string msg("Could not allocate device memory for VectorGpu: ");
            msg += hipGetErrorString(cudaResult);
            throw std::runtime_error(msg);
        }

        //std::cout << "VectorGpu(size_t size): Device memory for VectorGpu allocated!\n";
    }

    VectorGpu(VectorCpu<T> vecCpu) : _size(vecCpu.GetSize())
    {
        std::cout << "VectorGpu(VectorCpu<T> vecCpu) constructor started...\n";

        if (_size == 0)
        {
            std::string mes = "Cannot initialize vector of _size = 0";
            //std::cerr << mes << std::endl;
            throw std::logic_error(mes);
        }

        hipError_t cudaResult = hipMalloc(&_dev_data, _size*sizeof(T));
        if (cudaResult != hipSuccess)
        {
            std::string msg("Could not allocate device memory for VectorGpu: ");
            msg += hipGetErrorString(cudaResult);
            throw std::runtime_error(msg);
        }

        std::cout << "VectorGpu(VectorCpu<T> vecCpu): Device memory for VectorGpu allocated!\n";
    
        // Копируем данные в видеопамять
        cudaResult = hipMemcpy(_dev_data, vecCpu.Get_data_pointer(), _size*sizeof(T), hipMemcpyHostToDevice);
        if (cudaResult != hipSuccess)
        {
            std::string msg("Could not copy data from RAM to device memory: ");
            msg += hipGetErrorString(cudaResult);
            throw std::runtime_error(msg);
        }
        //std::cout << "cudaMemCpy OK!\n";

        // Устанавливаем флаг инициализации вектора
        _isInitialized = true;
    
    }

    ~VectorGpu()
    {
        //std::cout << "~VectorGpu(): " << this << " destructed!\n";
    }

    /// @brief Проверяет состояние вектора
    bool CheckState()
    {
        if(!_isInitialized)
            return false;

        if(_size < 1)
            return false;

        if(_dev_data == nullptr)
            return false;

        return true;
    }

    /// @brief Возвращает сумму элементов вектора
    FuncResultScalar<T> Sum(unsigned blocksNum, unsigned threadsNum)
    {
        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

        if(!CheckState())
            throw std::logic_error("Vector is not initialized!");      

        T result = CudaHelper<T>::Sum(_dev_data, _size, blocksNum, threadsNum);       

        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        //std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[us]" << std::endl;

        FuncResultScalar<T> res{true, result, std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()};
        //res.Print();

        return res;
    }

    /// @brief Освобождаем массив в видеопамяти
    void Clear_dev_data()
    {
        if(_dev_data != nullptr)
        {
            hipFree(_dev_data);
            _dev_data = nullptr;
            _isInitialized = false;
            //std::cout << "Device memory for VectorGpu cleared!\n";
        }
    }

    /// @brief Возвращает указатель на данные в видеопамяти
    /// @return 
    __host__ __device__
    T* Get_dev_data_pointer()
    {
        return _dev_data;
    }

    __host__ __device__
    size_t GetSize() const
    {
        return _size;
    }
    
    /// @brief Инициализирует вектор числом
    void InitVectorByScalar(double value)
    {
        // Создаём временный массив
        T* tmp = new T[_size];
        
        // Инициализируем временный массив        
        for (auto i = 0; i < _size; i++)
        {
            tmp[i] = value;
            //std::cout << tmp[i] << " ";
        }
        //std::cout << std::endl;

        // Копируем данные из временного массива в видеопамять
        hipError_t cudaResult = hipMemcpy(_dev_data, tmp, _size*sizeof(T), hipMemcpyHostToDevice);
        if (cudaResult != hipSuccess)
        {
            std::string msg("Could not copy data from RAM to device memory: ");
            msg += hipGetErrorString(cudaResult);
            throw std::runtime_error(msg);
        }

        //std::cout << "cudaMemCpy OK!\n";

        // Освобождаем временный массив
        delete[] tmp;

        // Устанавливаем флаг инициализации вектора
        _isInitialized = true;
    }

    /// @brief Инициализирует вектор числами из диапазона от start до end
    void InitVectorByRange(double start, double end)
    {
        // Создаём временный массив
        T* tmp = new T[_size];
        size_t cnt = 0;

        // Инициализируем временный массив
        auto step = (end-start)/(_size-1);
        for (auto i = start; i < end+step/2; i+=step)
        {
            tmp[cnt++] = i;
            //std::cout << tmp[cnt-1] << " ";
        }
        std::cout << std::endl;

        // Копируем данные из временного массива в видеопамять
        hipError_t cudaResult = hipMemcpy(_dev_data, tmp, _size*sizeof(T), hipMemcpyHostToDevice);
        if (cudaResult != hipSuccess)
        {
            std::string msg("Could not copy data from RAM to device memory: ");
            msg += hipGetErrorString(cudaResult);
            throw std::runtime_error(msg);
        }

        //std::cout << "cudaMemCpy OK!\n";

        // Освобождаем временный массив
        delete[] tmp;

        // Устанавливаем флаг инициализации вектора
        _isInitialized = true;
    }

    void Print()
    {
        print_kernel<T><<<1,1>>>(_dev_data, 0, _size);
        hipDeviceSynchronize();
    }
};