#pragma once

#include <iostream>
#include "CudaHelper.cu"

/// @brief Вектор (в GPU) 
/// @tparam T Тип элементов вектора
template<typename T = double>
class VectorGpu
{
    // Количество элементов вектора
    size_t _size = 0;
    // Указатель на массив в видеопамяти
    T* _dev_data = nullptr;
    // Флаг инициализации вектора
    // false - неинициализирован, true - инициализирован
    bool _isInitialized = false;

public:
    VectorGpu(size_t size) : _size(size)
    {
        std::cout << "VectorGpu(size_t size) constructor started...\n";

        if (_size == 0)
        {
            std::string mes = "Cannot initialize vector of _size = 0";
            //std::cerr << mes << std::endl;
            throw std::logic_error(mes);
        }

        hipError_t cudaResult = hipMalloc(&_dev_data, size*sizeof(T));
        if (cudaResult != hipSuccess)
        {
            std::string msg("Could not allocate device memory for VectorGpu: ");
            msg += hipGetErrorString(cudaResult);
            throw std::runtime_error(msg);
        }

        std::cout << "Device memory for VectorGpu allocated!\n";
    }

    ~VectorGpu()
    {
        std::cout << "~VectorGpu(): " << this << " destructed!\n";
    }

    /// @brief Проверяет состояние вектора
    bool CheckState()
    {
        if(!_isInitialized)
            return false;

        if(_size < 1)
            return false;

        if(_dev_data == nullptr)
            return false;

        return true;
    }

    /// @brief Возвращает сумму элементов вектора
    T Sum(unsigned blocksNum, unsigned threadsNum)
    {
        if(!CheckState())
            throw std::logic_error("Vector is not initialized!");      

        T result = CudaHelper<T>::Sum(_dev_data, _size, blocksNum, threadsNum);

        return result;
    }

    /// @brief Освобождаем массив в видеопамяти
    void Clear_dev_data()
    {
        if(_dev_data != nullptr)
        {
            hipFree(_dev_data);
            _dev_data = nullptr;
            _isInitialized = false;
            std::cout << "Device memory for VectorGpu cleared!\n";
        }
    }

    /// @brief Возвращает указатель на данные в видеопамяти
    /// @return 
    __host__ __device__
    T* get_dev_data_pointer()
    {
        return _dev_data;
    }

    __host__ __device__
    size_t getSize() const
    {
        return _size;
    }
    
    void initVectorByRange(double start, double end)
    {
        // Создаём временный массив
        T* tmp = new T[_size];
        size_t cnt = 0;

        // Инициализируем временный массив
        auto step = (end-start)/(_size-1);
        for (auto i = start; i < end+step/2; i+=step)
        {
            tmp[cnt++] = i;
            std::cout << tmp[cnt-1] << " ";
        }
        std::cout << std::endl;

        // Копируем данные из временного массива в видеопамять
        hipError_t cudaResult = hipMemcpy(_dev_data, tmp, _size*sizeof(T), hipMemcpyHostToDevice);
        if (cudaResult != hipSuccess)
        {
            std::string msg("Could not copy data from RAM to device memory: ");
            msg += hipGetErrorString(cudaResult);
            throw std::runtime_error(msg);
        }

        std::cout << "cudaMemCpy OK!\n";

        // Освобождаем временный массив
        delete[] tmp;

        // Устанавливаем флаг инициализации вектора
        _isInitialized = true;
    }

};